#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

__global__ void conv1d(int *input, int *kernel, int *output, int l, int k) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int r = k / 2;
	int start = tid - r;
	int temp = 0;
	for (int j = 0; j < k; j++) {
		if ((start + j >= 0) && (start + j < l)) {
			temp += input[start + j] * kernel[j];
		}
	}
	output[tid] = temp;
}

int main() {
	int l = 20480;
	int k = 7;
	int i;
	int *input, *kernel, *output;
	int *dev_input, *dev_kernel, *dev_output;

	hipMalloc((void**)&dev_input, sizeof(int) * l);
	hipMalloc((void**)&dev_kernel, sizeof(int) * k);
	hipMalloc((void**)&dev_output, sizeof(int) * l);
	hipHostMalloc((void**)&input, sizeof(int) * l, hipHostMallocDefault);
	hipHostMalloc((void**)&kernel, sizeof(int) * k, hipHostMallocDefault);
	hipHostMalloc((void**)&output, sizeof(int) * l, hipHostMallocDefault);

	for (i = 0; i < l; i++) {
		input[i] = round(rand());
	}
	for (i = 0; i < k; i++) {
		kernel[i] = round(rand());
	}

	printf("Start convolution\n");
	clock_t start_time = clock();
	hipMemcpy(dev_input, input, sizeof(int) * l, hipMemcpyHostToDevice);
	hipMemcpy(dev_kernel, kernel, sizeof(int) * k, hipMemcpyHostToDevice);

	int block = 256;
	int grid = (l + block - 1) / block;
	conv1d<<<grid, block>>>(input, kernel, output, l, k);

	hipMemcpy(output, dev_output, sizeof(int) * l, hipMemcpyDeviceToHost);
	clock_t end_time = clock();
	printf("Time consuming of 1D convolution of %d array with %d kernel is %f ms.\n", l, k, static_cast<double>(end_time - start_time)/CLOCKS_PER_SEC*1000);

	hipFree(dev_input);
	hipFree(dev_kernel);
	hipFree(dev_output);
	hipHostFree(input);
	hipHostFree(kernel);
	hipHostFree(output);

	return 0;
}
