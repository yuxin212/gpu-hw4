#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

#define KERNEL_SIZE 20

__constant__ int kernel[KERNEL_SIZE];

__global__ void conv1d(int *input, int *output, int l) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int r = KERNEL_SIZE / 2;
	int start = tid - r;
	int temp = 0;
	for (int j = 0; j < KERNEL_SIZE; j++) {
		if ((start + j >= 0) && (start + j < l)) {
			temp += input[start + j] * kernel[j];
		}
	}
	output[tid] = temp;
}

int main() {
	int l = 20480;
	int i;
	int *host_input, *host_kernel, *host_output;
	int *dev_input, *dev_output;

	hipMalloc((void**)&dev_input, sizeof(int) * l);
	hipMalloc((void**)&dev_output, sizeof(int) * KERNEL_SIZE);
	hipHostMalloc((void**)&host_input, sizeof(int) * l, hipHostMallocDefault);
	hipHostMalloc((void**)&host_kernel, sizeof(int) * KERNEL_SIZE, hipHostMallocDefault);
	hipHostMalloc((void**)&host_output, sizeof(int) * l, hipHostMallocDefault);

	for (i = 0; i < l; i++) {
		host_input[i] = round(rand());
	}
	for (i = 0; i < KERNEL_SIZE; i++) {
		host_kernel[i] = round(rand());
	}

	clock_t start_time = clock();
	hipMemcpy(dev_input, host_input, sizeof(int) * l, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(kernel), host_kernel, sizeof(int) * KERNEL_SIZE);

	int block = 256;
	int grid = (l + block - 1) / block;

	conv1d<<<grid, block>>>(dev_input, dev_output, l);

	hipMemcpy(host_output, dev_output, sizeof(int) * l, hipMemcpyDeviceToHost);
	clock_t end_time = clock();
	printf("Time consuming of 1D convolution of %d array with %d kernel is %f ms.\n", l, KERNEL_SIZE, static_cast<double>(end_time - start_time)/CLOCKS_PER_SEC*1000);

	hipFree(dev_input);
	hipFree(dev_output);
	hipHostFree(host_input);
	hipHostFree(host_kernel);
	hipHostFree(host_output);

	return 0;
}
